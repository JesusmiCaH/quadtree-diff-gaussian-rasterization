#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"


// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}


__device__ bool BoxInsideGaussian(
	uint2 rect_min, uint2 rect_max,
	float2 points_xy,
	float3 cov2D_inv,
	float confidence,
)
{
	// Transform the rectangle corners relative to the Gaussian center
	int2 c_rect_min = { rect_min.x - points_xy.x, rect_min.y - points_xy.y };
	int2 c_rect_max = { rect_max.x - points_xy.x, rect_max.y - points_xy.y };

	// Check each corner of the rectangle
	float corners[4][2] = {
		{ c_rect_min.x, c_rect_min.y },
		{ c_rect_max.x, c_rect_min.y },
		{ c_rect_min.x, c_rect_max.y },
		{ c_rect_max.x, c_rect_max.y }
	};

	uint n_inside = 0;

	for (int i = 0; i < 4; i++) {
		float x = corners[i][0];
		float y = corners[i][1];


		// Evaluate the ellipse equation: cov2D.x * x^2 + 2 * cov2D.y * x * y + cov2D.z * y^2 <= 1
		float ellipse_value = cov2D_inv.x * x * x + 2.0f * cov2D_inv.y * x * y + cov2D_inv.z * y * y;
		// printf("ellipse value is %f\n", ellipse_value);
		if (ellipse_value < confidence) {
			n_inside++;
			if(n_inside>=2) return true;
		}
	}
	// All corners are inside the ellipse
	return false;
}

__device__ bool GaussianInsideBox(
	uint2 rect_min, uint2 rect_max,
	float2 points_xy)
{
	return (points_xy.x >= rect_min.x && points_xy.x < rect_max.x &&
		points_xy.y >= rect_min.y && points_xy.y < rect_max.y);
}

__device__ bool GaussianIntersectBox(
	uint2 rect_min, uint2 rect_max,
	float2 points_xy,
	float3 cov2D_inv,
	float tolerant,
)
{
	float2 c_rect_min = { rect_min.x, rect_min.y };
	float2 c_rect_max = { rect_max.x, rect_max.y };

	c_rect_max.x -= points_xy.x;
	c_rect_max.y -= points_xy.y;
	c_rect_min.x -= points_xy.x;
	c_rect_min.y -= points_xy.y;

	
	
	float delta_n = c_rect_min.y * c_rect_min.y * cov2D_inv.y * cov2D_inv.y - cov2D_inv.x * ( cov2D_inv.z * c_rect_min.y * c_rect_min.y -1);
	if (delta_n >= 0){
		float x_min = (- cov2D_inv.y * c_rect_min.y - sqrt(delta_n)) / cov2D_inv.x;
		float x_max = (- cov2D_inv.y * c_rect_min.y + sqrt(delta_n)) / cov2D_inv.x;
		if (x_min < c_rect_max.x+tolerant && x_max > c_rect_min.x-tolerant)
			return true;
	}

	float delta_s = c_rect_max.y * c_rect_max.y * cov2D_inv.y * cov2D_inv.y - cov2D_inv.x * ( cov2D_inv.z * c_rect_max.y * c_rect_max.y -1);
	if (delta_s >= 0){
		float x_min = (- cov2D_inv.y * c_rect_max.y - sqrt(delta_s)) / cov2D_inv.x;
		float x_max = (- cov2D_inv.y * c_rect_max.y + sqrt(delta_s)) / cov2D_inv.x;
		if (x_min < c_rect_max.x+tolerant && x_max > c_rect_min.x-tolerant)
			return true;
	}

	float delta_w = c_rect_min.x * c_rect_min.x * cov2D_inv.y * cov2D_inv.y - cov2D_inv.z * ( cov2D_inv.x * c_rect_min.x * c_rect_min.x -1);
	if (delta_w >= 0){
		float y_min = (- cov2D_inv.y * c_rect_min.x - sqrt(delta_w)) / cov2D_inv.z;
		float y_max = (- cov2D_inv.y * c_rect_min.x + sqrt(delta_w)) / cov2D_inv.z;
		if (y_min < c_rect_max.y+tolerant && y_max > c_rect_min.y-tolerant)
			return true;
	}

	float delta_e = c_rect_max.x * c_rect_max.x * cov2D_inv.y * cov2D_inv.y - cov2D_inv.z * ( cov2D_inv.x * c_rect_max.x * c_rect_max.x -1);
	if (delta_e >= 0){
		float y_min = (- cov2D_inv.y * c_rect_max.x - sqrt(delta_e)) / cov2D_inv.z;
		float y_max = (- cov2D_inv.y * c_rect_max.x + sqrt(delta_e)) / cov2D_inv.z;
		if (y_min < c_rect_max.y+tolerant && y_max > c_rect_min.y-tolerant)
			return true;
	}
	
	return false;
}

#define MAX_STACK 64

__device__ void kvpairByQuadtree(
	uint2 rect_min, uint2 rect_max,
	dim3 grid,
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,
	uint32_t offset,
	uint32_t idx,
	float2 points_xy,
	const float* depths,
	float3 cov2D,
	float tolerant,
	float confidence
)
{
	// 模拟栈
	uint2 stack_min[MAX_STACK];
	uint2 stack_max[MAX_STACK];
	int stack_ptr = 0;

	// 初始压入
	stack_min[stack_ptr] = rect_min;
	stack_max[stack_ptr] = rect_max;
	stack_ptr++;

	while (stack_ptr > 0)
	{
		stack_ptr--;
		uint2 cur_min = stack_min[stack_ptr];
		uint2 cur_max = stack_max[stack_ptr];

		uint2 real_rect_min = {cur_min.x * BLOCK_X, cur_min.y * BLOCK_Y};
		uint2 real_rect_max = {cur_max.x * BLOCK_X, cur_max.y * BLOCK_Y};
		// Compute the inverse of the 2D covariance matrix
		float det = cov2D.x * cov2D.z - cov2D.y * cov2D.y;
		float3 cov2D_inv = {
			cov2D.z / det,
			-cov2D.y / det,
			cov2D.x / det
		};

		if (
			GaussianInsideBox(real_rect_min, real_rect_max, points_xy) ||
			GaussianIntersectBox(real_rect_min, real_rect_max, points_xy, cov2D_inv, tolerant) ||
			BoxInsideGaussian(real_rect_min, real_rect_max, points_xy, cov2D_inv, confidence) ||
			false
		)
		{
			if (cur_max.x - cur_min.x == 0 || cur_max.y - cur_min.y == 0)
			{
				continue;
			}
			else if (cur_max.x - cur_min.x == 1 && cur_max.y - cur_min.y == 1)
			{
				uint64_t key = cur_min.y * grid.x + cur_min.x;
				key <<= 32;
				key |= *((uint32_t*)&depths[idx]);
				gaussian_keys_unsorted[offset] = key;
				gaussian_values_unsorted[offset] = idx;
				offset++;
			}
			else if (stack_ptr + 4 <= MAX_STACK)
			{
				uint2 mid = {
					(cur_min.x + cur_max.x) / 2,
					(cur_min.y + cur_max.y) / 2};

				// 四个象限压入栈
				stack_min[stack_ptr] = cur_min;
				stack_max[stack_ptr] = mid;
				stack_ptr++;

				stack_min[stack_ptr] = {mid.x, cur_min.y};
				stack_max[stack_ptr] = {cur_max.x, mid.y};
				stack_ptr++;

				stack_min[stack_ptr] = {cur_min.x, mid.y};
				stack_max[stack_ptr] = {mid.x, cur_max.y};
				stack_ptr++;

				stack_min[stack_ptr] = mid;
				stack_max[stack_ptr] = cur_max;
				stack_ptr++;
			}
		}
	}
}

// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,
	int* radii,
	float3* cov2Ds,
	dim3 grid,
	float tolerant,
	float confidence)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

		uint32_t off_tree = off;

		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// printf("idx %lld, offset %d \n", idx, off);
		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 

		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				// uint64_t key = y * grid.x + x;
				// printf("key is %llu, value is %d\n", key, y * grid.x + x);
				// key <<= 32;
				// key |= *((uint32_t*)&depths[idx]);
				// gaussian_keys_unsorted[off] = key;
				gaussian_keys_unsorted[off] = 0;
				// gaussian_values_unsorted[off] = idx;
				// printf("offset = %d, gaussiankeysunsorted = %lld \n", off, gaussian_keys_unsorted[off]);
				off++;	
			}

		}


		// Use quadtree to find the tile that the Gaussian intersects
		// and add the key/value pair to the list.
		kvpairByQuadtree(
			rect_min, rect_max,
			grid,
			gaussian_keys_unsorted,
			gaussian_values_unsorted,
			off_tree,
			idx,
			points_xy[idx],
			depths,
			cov2Ds[idx],
			tolerant,
			confidence,
		);

	}
}


// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();

	if (idx >= L){
		return;
	}

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	if (key==0){
		return;
	}
	uint32_t currtile = key >> 32;
	uint32_t prevtile = point_list_keys[idx - 1] >> 32;

	// printf("IDX %lld, istrue? %d \n", idx, currtile != prevtile);

	if (idx == 0 || point_list_keys[idx - 1] == 0){
		ranges[currtile].x = idx;
	}
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;

		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
	// printf("range = %d, %d\n", ranges[currtile].x, ranges[currtile].y);
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color,
	float* depth,
	bool antialiasing,
	int* radii,
	bool debug,
	float tolerant,
	float confidence)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	size_t chunk_size = required<GeometryState>(P);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	size_t img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	float3* cov2Ds;
	CHECK_CUDA(hipMalloc((void**)&cov2Ds, P * sizeof(float3)), debug);

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,
		geomState.means2D,
		geomState.depths,
		geomState.cov3D,
		cov2Ds,
		geomState.rgb,
		geomState.conic_opacity,
		tile_grid,
		geomState.tiles_touched,
		prefiltered,
		antialiasing
	), debug)

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

	printf("numrenderd %d\n", num_rendered);

	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		cov2Ds,
		tile_grid,
		tolerant,

	);
	
	hipDeviceSynchronize();

	CHECK_CUDA(hipGetLastError(), debug)

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)
	
	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

	// // Debugging: Check all the values inside binningState.point_list_keys
	// std::vector<uint64_t> h_point_list_keys(num_rendered);
	// hipMemcpy(h_point_list_keys.data(), binningState.point_list_keys, num_rendered * sizeof(uint64_t), hipMemcpyDeviceToHost);

	// for (int i = 0; i < num_rendered; ++i) {
	// 	printf("point_list_keys[%d] = %llu\n", i, h_point_list_keys[i]>>32);
	// }

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)

	// Debugging: Check the imgState.ranges
	std::vector<uint2> h_ranges(tile_grid.x * tile_grid.y);
	hipMemcpy(h_ranges.data(), imgState.ranges, tile_grid.x * tile_grid.y * sizeof(uint2), hipMemcpyDeviceToHost);

	for (int i = 0; i < tile_grid.x * tile_grid.y; ++i) {
		printf("不应如是");
		printf("Tile %d: range start = %u, range end = %u\n", i, h_ranges[i].x, h_ranges[i].y);
	}

	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	CHECK_CUDA(FORWARD::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		geomState.means2D,
		feature_ptr,
		geomState.conic_opacity,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		out_color,
		geomState.depths,
		depth), debug)

	return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	const float* dL_invdepths,
	float* dL_dmean2D,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dinvdepth,
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot,
	bool antialiasing,
	bool debug)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		geomState.depths,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		dL_invdepths,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor,
		dL_dinvdepth), debug);

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		opacities,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		dL_dinvdepth,
		dL_dopacity,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot,
		antialiasing), debug);
}
